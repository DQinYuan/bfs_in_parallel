#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by CUDA without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <deque>
#include <sys/time.h>


#define BLOCK_IN_A_GRID 80
#define THREAD_PER_BLOCK 64
#define MAX_THREAD_PER_BLOCK 256


/*
 the method is a little different with the naive one

 it uses two current sets

 */

__global__ static void bfs_kernel(unsigned int* current_set, unsigned int* new_set,
                                  int* current_set_size, int* current_set_size_new,
                                  Node* node_list, Edge* edge_list, int* color, int* cost, int level)
{
	
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
     *current_set_size_new = 0;  // at first the current set must be empty
     
     if (tid < current_set_size) {  // the rest threads take a rest

          unsigned int index = current_set[tid];// fetch one from the current set
          current_set[tid] = 0;                 // erase it        
          
          d_cost[index] = level;

          Node cur_node = node_list[index];
          for (int i = cur_node.start; i < cur_node.start + cur_node.edge_num; i++)
          {
               unsigned int id = edge_list[i].dest;

               // use the atomic operation to prevent confliction
               // there is only one chance to increase the length
               // current set
               int its_color = atomicExch((int*) &color[id], BLACK);
               if (its_color == WHITE) {
                    int write_position = atomicAdd((int*) &(*current_set_size_new), 1);
                    new_set[write_position] = id;
               }
          }
          
     }
}


float bfs() 
{
	 struct timeval start, end;
	 float time_used;
	 gettimeofday(&start, 0);

	 // visiting the source node now(CPU)
	 color[source_node_no] = BLACK;
	 current_set[0]= source_node_no;
	 cost[source_node_no] = 0;
	 
     // synchronize to GPU mem
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice);
     cudaMemcopy(d_current_set_a, current_set, sizeof(unsigned int) * num_of_nodes,
                 hipMemcpyHostToDevice);
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice); 
     int current_set_size = 1;          // only source node in it     
     int block_num = BLOCK_IN_A_GRID;
     int thread_num = THREAD_PER_BLOCK;

     int level = 0;                     // used to control the current_set_a/b to visit
	 while(current_set_size != 0) {
          if (level%2 == 0) {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_a, d_current_set_b,
                                                     current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               // update the size of current_size after adding nodes to current set
               hipMemcpy(current_set_size_new, d_current_set_size_new,
                          sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; 
          } else {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_b, d_current_set_a,
                                                     current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               // update the size of current_size after adding nodes to current set
               hipMemcpy(current_set_size_new, de_current_set_size_new,
                          sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new;
          }
          level++;
	 }

     // copy the result from GPU to CPU mem
     cudaMemcopy(cost, d_cost, sizeof(unsigned int)*num_of_nodes, hipMemcpyDeviceToHost);

     // come out the time
	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
