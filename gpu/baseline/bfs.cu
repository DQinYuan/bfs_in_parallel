#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by OpenMP without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <deque>
#include <sys/time.h>

__global__ static void kernel() {
}


float bfs(void) 
{
	 struct timeval start, end;
	 float time_used;
	 std::deque<unsigned int> current;

	 gettimeofday(&start, 0);

	 // visiting the source node now
	 color[source_node_no] = GREY;
	 current.push_back(source_node_no);
	 cost[source_node_no] = 0;
	 
	 unsigned int index;
	 while(!current.empty()) {
		  
		  for (int k=0; k<current.size(); k++) {

			   // pop out a node to deal with
			   index = current.front();
			   current.pop_front();

			   // put all its neighbours in the current queue
			   for (int i = node_list[index].start;
					i < (node_list[index].start + node_list[index].edge_num);
					i ++) {
					unsigned int id = edge_list[i].dest;
					//unsigned int weight = edge_list[i].cost;
			   
					if (color[id] == WHITE) {
						 cost[id] = cost[index] + 1; // expand the cost, assuming all the edge cost is 1
						 counter[cost[id]] ++;
						 current.push_back(id);
						 color[id] = GREY;

					} // only if its neighbour is has not been visited
			   }
			   color[index] = BLACK;
		  }
	 }
	 
	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
