#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by CUDA without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <deque>
#include <sys/time.h>


#define BLOCK_IN_A_GRID 80
#define THREAD_PER_BLOCK 64
#define MAX_THREAD_PER_BLOCK 256

__global__ static void bfs_kernel(unsigned int* current_set, unsigned int* new_set)
{
	
     int tid = blockIdx.x * MAX_THREAD_PER_BLOCK + threadIdx.x;
     if (tid<num_of_nodes && d_color[tid] == WHITE ) {
          
     }


}


float bfs() 
{
	 struct timeval start, end;
	 float time_used;
	 gettimeofday(&start, 0);

	 // visiting the source node now(CPU)
	 color[source_node_no] = GREY;
	 current_set[0]= source_node_no;
	 cost[source_node_no] = 0;
	 
     // synchronize to GPU mem
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes,
				 hipMemcpyHostToDevice);
     cudaMemcopy(d_current_set_a, current_set, sizeof(unsigned int) * num_of_nodes,
                 hipMemcpyHostToDevice);
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes,
				 hipMemcpyHostToDevice);
          
     int current_set_size = 1;          // only source node in it 
     int level = 0;             // used to mark the current_set_a/b to visit
     int block_num = BLOCK_IN_A_GRID;
     int thread_num = THREAD_PER_BLOCK;
     
	 while(current_set_size != 0) {
          if (level%2 == 0) {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_a, d_current_set_b);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int)*1, hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; // update the set saize
          } else {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_b, d_current_set_a);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, de_current_set_size_new, sizeof(int)*1, hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new;
          }
          level++;
	 }
	 
	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
