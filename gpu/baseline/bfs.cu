#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by CUDA without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <sys/time.h>

#define MAX_THREAD_PER_BLOCK 1024
#define THREAD_PER_BLOCK 128

/*
 the method is a little different with the naive one

 it uses two current sets

 */




__global__ static void bfs_kernel(unsigned int* current_set, unsigned int* new_set,
                                  int current_set_size, int* current_set_size_new, 
                                  Node* node_list, Edge* edge_list, int* color, int* cost, int level)
{
	
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
     
     for(int j=tid; j<current_set_size; j+=blockDim.x*gridDim.x) {
          unsigned int index = current_set[j];// fetch one from the current set
          current_set[j] = 0;                 // erase it
          cost[index] = level;
          Node cur_node = node_list[index];
          for (int i=cur_node.start; i < cur_node.start + cur_node.edge_num; i++)
          {
               unsigned int id = edge_list[i].dest;
               int its_color = atomicExch((int*) &color[id], BLACK);
               if (its_color == WHITE) {
                    int write_position = atomicAdd((int*) &(*current_set_size_new), 1);
                    new_set[write_position] = id;
                   
                    
               }
          }
     }
}


float bfs(int block_in_a_grid) 
{
	 struct timeval start, end;
	 float time_used;
	 gettimeofday(&start, 0);

	 // visiting the source node now(CPU)
	 color[source_node_no] = BLACK;
	 current_set[0]= source_node_no;
	 cost[source_node_no] = 0;
	 
     // synchronize to GPU mem
     hipMemcpy(d_color, color, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice);
     hipMemcpy(d_current_set_a, current_set, sizeof(unsigned int) * num_of_nodes, hipMemcpyHostToDevice);
     hipMemcpy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice); 


     //hipMemset(d_color, BLACK, sizeof(int));
     //hipMemset(d_current_set_a[0], source_node_no, sizeof(unsigned int));
     //hipMemset(d_cost[source_node_no], 0, sizeof(int));


     int current_set_size = 1;          // only source node in it     
     int block_num = block_in_a_grid;
     int thread_num = THREAD_PER_BLOCK;


     int level = 0;                     // used to control the current_set_a/b to visit
	 while(current_set_size != 0) {
          if (level%2 == 0) {
               hipMemset(d_current_set_size_new, 0, sizeof(int));
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_a, d_current_set_b, current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; 
               
          } else {

               hipMemset(d_current_set_size_new, 0, sizeof(int));
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_b, d_current_set_a, current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new;
               
          }
          level++;
	 }

     // copy the result from GPU to CPU mem
     hipMemcpy(cost, d_cost, sizeof(unsigned int)*num_of_nodes, hipMemcpyDeviceToHost);

     // come out the time
	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
