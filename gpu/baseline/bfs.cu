#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by CUDA without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <deque>
#include <sys/time.h>


#define BLOCK_IN_A_GRID 80
#define THREAD_PER_BLOCK 64
#define MAX_THREAD_PER_BLOCK 256

__global__ static void bfs_kernel(unsigned int* current_set, unsigned int* new_set, int* current_set_size,
                                  int* cost, Node* node_list, Edge* edge_list, int* color, int* level, int* current_set_size_new)
{
	
     int tid = blockIdx.x*blockDim.x + threadIdx.x;
     if (tid < current_set_size) {  // the rest threads take a rest
          unsigned int index = current_set[tid];  // fetch one from the current set
          
          d_cost[index] = level;

          for (int i = node_list[index].start;
               i < (node_list[index].start + node_list[index].edge_num);
               i++)
          {
               unsigned int id = edge_list[i].dest;
               int 
          }
          
          
          
     }


}


float bfs() 
{
	 struct timeval start, end;
	 float time_used;
	 gettimeofday(&start, 0);

	 // visiting the source node now(CPU)
	 color[source_node_no] = GREY;
	 current_set[0]= source_node_no;
	 cost[source_node_no] = 0;
	 
     // synchronize to GPU mem
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice);
     cudaMemcopy(d_current_set_a, current_set, sizeof(unsigned int) * num_of_nodes,
                 hipMemcpyHostToDevice);
     cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice); 
     int current_set_size = 1;          // only source node in it 
     int level = 0;                     // used to control the current_set_a/b to visit
     int block_num = BLOCK_IN_A_GRID;
     int thread_num = THREAD_PER_BLOCK;
     
	 while(current_set_size != 0) {
          if (level%2 == 0) {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_a, d_current_set_b, current_set_size, d_cost,
                                                     d_node_list, d_edge_list, d_color, level, d_current_set_size_new);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int)*1, hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; // update the size of current_size
          } else {
               bfs_kernel<<<block_num, thread_num>>>(d_current_set_b, d_current_set_a, current_set_size, d_cost,
                                                     d_node_list, d_edge_list, d_color, level, d_current_set_size_new);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, de_current_set_size_new, sizeof(int)*1, hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; // update the size of current_size
          }
          level++;
	 }

     //copy the result from GPU to CPU mem
     cudaMemcopy(cost, d_cost, sizeof(unsigned int)*num_of_nodes, hipMemcpyDeviceToHost);

	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
