/************************************************************************
filename: gpu/cuda.cu
author: onesuper
email: onesuperclark@gmail.com

allocate and free memory on device
for common use
*************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

bool init_cuda() 
{
	int count = 0;

	hipGetDeviceCount(&count);
	
	printf("%d device\n", count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i=0; i<count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}


void device_alloc_and_copy(void) {

	 hipMalloc((void**) &d_node_list, sizeof(Node) * num_of_nodes);
	 hipMemcpy(d_nodes_list, node_list, sizeof(Node) * num_of_nodes,
				hipMemcpyHostToDevice);

	 hipMalloc((void**) &d_edge_list, sizeof(Edge) * num_of_edges);
	 cudaMemcopy(d_edge_list, edge_list. sizeof(Edge) * num_of_edges,
				 hipMemcpyHostToDevice);

	 hipMalloc((void**) &d_color, sizeof(int) * num_of_nodes);
	 cudaMemcopy(d_color, color, sizeof(int) * num_of_nodes,
				 hipMemcpyHostToDevice);

	 hipMalloc((void**) &d_cost, sizeof(int) * num_of_nodes);
	 cudaMemcopy(d_cost, cost, sizeof(int) * num_of_nodes,
				 hipMemcpyHostToDevice);

	 hipMalloc((void**) &d_counter, sizeof(int) * MAX_LEVEL);
	 cudaMemcopy(d_counter, counter, sizeof(int) * MAX_LEVEL,
				 hipMemcpyHostToDevice);

	 return; 
}

void device_free(void) {
	 hipFree(d_node_list);
	 hipFree(d_edge_list);
	 hipFree(d_cost);
	 hipFree(d_color);
	 hipFree(d_counter);
	 return;
}
