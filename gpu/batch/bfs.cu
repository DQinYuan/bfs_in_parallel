#include "hip/hip_runtime.h"
/**********************************************************************
filename: gpu/baseline/bfs.cpp
author: onesuper
email: onesuperclark@gmail.com

bfs algorithm implemented by CUDA without any optimization.

***********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <sys/time.h>

#define MAX_THREAD_PER_BLOCK 1024
#define THREAD_PER_BLOCK 128
#define MAX_LOCAL_QUEUE_SIZE 1200
/*
 the method is a little different with the naive one

 it uses two current sets

 */




__global__ static void bfs_kernel(unsigned int* current_set, unsigned int* new_set,
                                  int current_set_size, int* current_set_size_new, 
                                  Node* node_list, Edge* edge_list, int* color, int* cost, int level)
{
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
     __shared__ int local_queue[MAX_LOCAL_QUEUE_SIZE];
     __shared__ int local_queue_size;
     __shared__ int new_set_cursor;

     __syncthreads();
     if (threadIdx.x == 0) {
          new_set_cursor = 0;
     }
     __syncthreads();
     
     
     for(int j=tid; j<(current_set_size+blockDim.x*gridDim.x); j+=blockDim.x*gridDim.x) {
          
          __syncthreads();
          if (threadIdx.x == 0) {
               local_queue_size = 0;
          }
          __syncthreads();

          if (j < current_set_size) {

               unsigned int index = current_set[j];// fetch one from the current set
               current_set[j] = 0;                 // erase it
               cost[index] = level;
               Node cur_node = node_list[index];
               for (int i=cur_node.start; i < cur_node.start + cur_node.edge_num; i++)
               {
                    unsigned int id = edge_list[i].dest;
                    int its_color = atomicExch((int*) &color[id], BLACK);
                    if (its_color == WHITE) {
                         int write_position = atomicAdd((int*) &local_queue_size, 1);
                         local_queue[write_position] = id;
                         //printf("push %d to local queue total:%d \n", id, local_queue_size);
                    }
               }
          }

          __syncthreads();
          
          if (threadIdx.x == 0) {
               new_set_cursor = atomicAdd((int*) &(*current_set_size_new), local_queue_size);
               // printf("new_set_cursor = %d\n", new_set_cursor);
          }
          __syncthreads();
          
          for (int i=threadIdx.x; i<local_queue_size; i+=blockDim.x) {
               new_set[new_set_cursor+i] = local_queue[i];
               //printf("push %d to next queue total:%d  \n",local_queue[i], *current_set_size_new);
          }
          __syncthreads();
          
     }

   




}


float bfs(int block_in_a_grid) 
{
	 struct timeval start, end;
	 float time_used;
	 gettimeofday(&start, 0);

	 // visiting the source node now(CPU)
	 color[source_node_no] = BLACK;
	 current_set[0]= source_node_no;
	 cost[source_node_no] = 0;
	 
     // synchronize to GPU mem
     hipMemcpy(d_color, color, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice);
     hipMemcpy(d_current_set_a, current_set, sizeof(unsigned int) * num_of_nodes, hipMemcpyHostToDevice);
     hipMemcpy(d_cost, cost, sizeof(int) * num_of_nodes, hipMemcpyHostToDevice); 


     //hipMemset(d_color, BLACK, sizeof(int));
     //hipMemset(d_current_set_a[0], source_node_no, sizeof(unsigned int));
     //hipMemset(d_cost[source_node_no], 0, sizeof(int));


     int current_set_size = 1;          // only source node in it     
     int block_num = block_in_a_grid;
     int thread_num = THREAD_PER_BLOCK;


     int level = 0;                     // used to control the current_set_a/b to visit
	 while(current_set_size != 0) {
          if (level%2 == 0) {
               hipMemset(d_current_set_size_new, 0, sizeof(int));
               bfs_kernel<<<block_num, thread_num, 10000>>>(d_current_set_a, d_current_set_b, current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new; 
               //printf("%d:%d\n",level,current_set_size);
               
          } else {

               hipMemset(d_current_set_size_new, 0, sizeof(int));
               bfs_kernel<<<block_num, thread_num, 10000>>>(d_current_set_b, d_current_set_a, current_set_size, d_current_set_size_new,
                                                     d_node_list, d_edge_list, d_color, d_cost, level);
               hipDeviceSynchronize();
               hipMemcpy(current_set_size_new, d_current_set_size_new, sizeof(int), hipMemcpyDeviceToHost);
               current_set_size = *current_set_size_new;
               //printf("%d:%d\n",level,current_set_size);
               
          }
          level++;
	 }

     // copy the result from GPU to CPU mem
     hipMemcpy(cost, d_cost, sizeof(unsigned int)*num_of_nodes, hipMemcpyDeviceToHost);

     // come out the time
	 gettimeofday(&end, 0);
	 time_used = 1000000 * (end.tv_sec - start.tv_sec) +
		  end.tv_usec - start.tv_usec;
	 time_used /= 1000000;
	 printf("used time: %f\n", time_used);
	 
	 return time_used;
	 
}
